#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <Windows.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <shlobj.h>
#include <>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <fstream>
#include <ctype.h>
#include "dicts.h"

#define LENGTH 72 // MAX LENGTH OF GENERATE NATIVE
#define MAX_LENGTH_WORD 16 // FOR ALLOC ( CHANGE IF A WORD IN DICT IS TALLER THAN 16 ), WILL BE A MULTIPLE OF 8
#define MIN_WORD_NUM 2 // MINIMUM OF WORDS TO GEN A NATIVE
#define MAX_WORD_NUM 6 // MAXIMUM OF WORDS TO GEN A NATIVE ( IF ERROR CHANGE THE LENGTH MACRO )

#define SIMULTANEOUS 200000 // SIMULTANEOUS NATIVES COUNT GENERATE PER CYCLE KERNEL
#define MAX_GENERATED 1000000 // MAX OUTPUT COLLISIONS

__global__ void initCURAND(hiprandState* cus, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
        hiprand_init(clock64(), i, 0, &(cus[i]));
}

__global__ void colliderKernel(hiprandState* cus, const char* dictStart, const char* dictArg,
                            const unsigned int* nativesUnk, unsigned short UnkNmb,
                            char* genNat, int numElements, int dictWordLength,
                            int dictStartWordLength)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
        char relOut[LENGTH];
        const unsigned char lengthWord64 = MAX_LENGTH_WORD / 8;
        uint64_t wordBuf[lengthWord64];

        const uint64_t* startWord =
            (const uint64_t*)(dictStart + ((unsigned char)(hiprand(&(cus[i])) % dictStartWordLength) * MAX_LENGTH_WORD));
        for (unsigned char it = 0; it < lengthWord64; it++)
            wordBuf[it] = startWord[it];
        const char* wordBuf8_1 = (const char*)wordBuf;
        
        unsigned char it1 = 0;
        while (wordBuf8_1[it1] != 0x00)
        {
            relOut[it1] = wordBuf8_1[it1];
            it1++;
        }
        relOut[it1] = '_';

        unsigned char randWords = (unsigned char)(hiprand(&(cus[i])) % (MAX_WORD_NUM - MIN_WORD_NUM)) + MIN_WORD_NUM; // BOUND 2 - 8
        for (unsigned char it2 = 0; it2 < randWords; it2++)
        {
            it1++;
            unsigned short randCurrent = (unsigned short)(hiprand(&(cus[i])) % dictWordLength);
            const uint64_t* current = (const uint64_t*)(dictArg + (randCurrent * MAX_LENGTH_WORD));
            for (unsigned char it = 0; it < lengthWord64; it++)
                wordBuf[it] = current[it];
            const char* wordBuf8_2 = (const char*)wordBuf;

            unsigned char itRel = 0;
            while (wordBuf8_2[itRel] != 0x00)
            {
                relOut[it1] = wordBuf8_2[itRel];
                it1++;
                itRel++;
            }
            relOut[it1] = '_';
        }
        relOut[it1] = 0x00;

        unsigned char it = 0;
        unsigned int hash = 0;
        while(relOut[it] != 0x00) {
            hash += relOut[it++];
            hash += hash << 10;
            hash ^= hash >> 6;
        }
        hash += hash << 3;
        hash ^= hash >> 11;
        hash += hash << 15;

        bool bypassErase = true;
        for (unsigned short it3 = 0; it3 < UnkNmb; it3++)
        {
            if (hash == nativesUnk[it3]) {
                bypassErase = false;
                break;
            }
        }
        
        if (bypassErase)
            *(genNat + (i * LENGTH)) = 0x00;
        else
        {
            int* relOutGlb = (int*)(genNat + (i * LENGTH));
            int* relOutFour = (int*)relOut;
            for (unsigned char itR = 0; itR < 18; itR++)
                if (relOutFour[itR] != 0x00000000)
                    relOutGlb[itR] = relOutFour[itR];
        }
    }
}

int main(void)
{
    hipDeviceProp_t propsDev = {};
    hipGetDeviceProperties(&propsDev, 0);
    hipSetDevice(0);
    printf("Selected GPU : %s\n\n", propsDev.name);

    int startWordsSize = sizeof(startWords) / sizeof(startWords[0]);
    char* dictTableStart = NULL;
    int sizeDict = startWordsSize * MAX_LENGTH_WORD;
    hipMalloc((void**)&dictTableStart, sizeDict);
    char* dictTableStartRAM = new char[sizeDict];
    memset(dictTableStartRAM, 0x00, sizeDict);

    for(int i = 0; i < startWordsSize; i++)
        memcpy(dictTableStartRAM + (i * MAX_LENGTH_WORD), startWords[i], strlen(startWords[i]));

    hipMemcpy(dictTableStart, dictTableStartRAM, sizeDict, hipMemcpyHostToDevice);
    delete[] dictTableStartRAM;


    int wordSizeDict = sizeof(words) / sizeof(words[0]);
    sizeDict = wordSizeDict * MAX_LENGTH_WORD;
    char* dictTable = NULL;
    hipMalloc((void**)&dictTable, sizeDict);
    char* dictTableRAM = new char[sizeDict];
    memset(dictTableRAM, 0x00, sizeDict);

    for(int i = 0; i < wordSizeDict; i++)
        memcpy(dictTableRAM + (i * MAX_LENGTH_WORD), words[i], strlen(words[i]));

    hipMemcpy(dictTable, dictTableRAM, sizeDict, hipMemcpyHostToDevice);
    delete[] dictTableRAM;

    printf("Dict start words size : %d entries\nDict words size : %d entries\n", startWordsSize, wordSizeDict);

    // RESULTS
    int sizeGen = SIMULTANEOUS * LENGTH;
    char* genPtr = NULL;
    hipMalloc((void**)&genPtr, sizeGen);
    char* genRAM = new char[sizeGen];
    memset(genRAM, 0x00, sizeGen);
    hipMemcpy(genPtr, genRAM, sizeGen, hipMemcpyHostToDevice);

    unsigned short UnknownsCount = sizeof(unknowns) / sizeof(unknowns[0]);
    unsigned int* UnknownsNat = NULL;
    int sizeNatUnk = UnknownsCount * 4;
    hipMalloc((void**)&UnknownsNat, sizeNatUnk);
    hipMemcpy((unsigned char*)UnknownsNat, (unsigned char*)unknowns, sizeNatUnk, hipMemcpyHostToDevice);

    int thrBloc = 128;
    int blocGrid = (SIMULTANEOUS + thrBloc - 1) / thrBloc;


    // CREATE OUTPUT TEXT
    TCHAR pathSP[1024];
	TCHAR pathSPO[1024];
    memset(pathSP, 0x00, 1024);
	memset(pathSPO, 0x00, 1024);
	SHGetSpecialFolderPath(HWND_DESKTOP, pathSP, CSIDL_DESKTOP, FALSE);
	strcpy_s(pathSPO, pathSP);
    strcat_s(pathSPO, "\\NativesGPU.txt");

    std::ofstream outputText = std::ofstream(pathSPO);
    char* outputBuf = new char[384];
    char* sprOutputBuf = new char[512];
    char* antiDouble = new char[LENGTH * MAX_GENERATED];


    hiprandState* cuRANDs;
    hipMalloc((void**)&cuRANDs, sizeof(hiprandState) * SIMULTANEOUS);
    initCURAND<<<blocGrid, thrBloc>>>(cuRANDs, SIMULTANEOUS);
    printf("Initialised CURAND\n\n");

    // STAT ABOUT
    int perSec = 0;
    int tickSec = GetTickCount();

    for(;;)
    {
        if (GetTickCount() - tickSec >= 1000)
        {
            printf("%d hashs/s\n", perSec);
            perSec = 0;
            tickSec = GetTickCount();
        }

        // EXECUTE HASH KERNEL
        colliderKernel<<<blocGrid, thrBloc>>>(cuRANDs, (const char*)dictTableStart, (const char*)dictTable,
                                                            (const unsigned int*)UnknownsNat, UnknownsCount,
                                                                    genPtr, SIMULTANEOUS, wordSizeDict, startWordsSize);

        hipMemcpy(genRAM, genPtr, sizeGen, hipMemcpyDeviceToHost);

        for (int i = 0; i < SIMULTANEOUS; i++)
        {
            char* Output = (genRAM + (i * LENGTH));
            if (*Output != 0x00)
            {
                bool bypassIA = true;
                for (int iA = 0; iA < MAX_GENERATED; iA++)
                {
                    char* CurIA = antiDouble + (iA * LENGTH);
                    if (*CurIA == 0x00)
                    {
                        memcpy(CurIA, Output, strlen(Output));
                        break;
                    }

                    if (!strcmp(Output, CurIA))
                    {
                        bypassIA = false;
                        break;
                    }
                }

                if (bypassIA)
                {
                    unsigned char it = 0;
                    unsigned int hash = 0;
                    while(Output[it] != 0x00) {
                        hash += Output[it++];
                        hash += hash << 10;
                        hash ^= hash >> 6;
                    }
                    hash += hash << 3;
                    hash ^= hash >> 11;
                    hash += hash << 15;

                    sprintf(sprOutputBuf, "0x%2X : %s", hash, Output);
                    unsigned short offset = 0;
                    while (sprOutputBuf[offset] != 0x00) {
                        if (offset != 1)
                            sprOutputBuf[offset] = toupper(sprOutputBuf[offset]);
                        offset++;
                    }

                    outputText << sprOutputBuf << std::endl;
                }
            }
        }

        perSec += SIMULTANEOUS;
    }

    /*
        GARBAGE COLLECTING
    */
    hipFree(cuRANDs);
    hipFree((char*)UnknownsNat);
    hipFree(genPtr);
    hipFree(dictTable);
    hipFree(dictTableStart);
    delete[] genRAM;
    outputText.close();

    return 0;
}
